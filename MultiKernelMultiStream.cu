#include "hip/hip_runtime.h"
#include "common/common.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#define NSTREAM 4

__device__ void kernel_func()
{
  double sum = 0.0;
  long i = 999999;
  while(i>0) {
    for (long j = 0; j < 999999; j++) {
      sum = sum + tan(0.1) * tan(0.1);
    }
    i = i - 1;
  }
}

__global__ void kernel(int stream, int id) {
  if (0 == threadIdx.x) {
    printf("kernel %d is executed in stream_%d\n", id, stream);
  }
  kernel_func();
}

int main(int argc, char**argv) {
  int nDeviceNumber = 0;
  // 检测当前设备与cuda兼容的设备
  ErrorCheck(hipGetDeviceCount(&nDeviceNumber));
  // set up device
  int dev = 0;
  // 设置GPU设备
  ErrorCheck(hipSetDevice(dev));

  float elapsed_time;

  int n_streams = NSTREAM;
  hipStream_t* streams = (hipStream_t*) malloc(n_streams * sizeof(hipStream_t));

  for (int i = 0; i< n_streams; i++) {
    ErrorCheck(hipStreamCreate(streams[i]));
  }

  dim3 block(1);
  dim3 grid(1);

  hipEvent_t start, stop;
  ErrorCheck(hipEventCreate(&start));
  ErrorCheck(hipEventCreate(&stop));

  // record start
  ErrorCheck(hipEventRecord(start, 0)); // 0表示走默认流方式

  for (int i = 0; i< n_streams; i++) {
    kernel<<<grid, block, 0, streams[i]>>>(i, 1);
    kernel<<<grid, block, 0, streams[i]>>>(i, 2);
    kernel<<<grid, block, 0, streams[i]>>>(i, 3);
    kernel<<<grid, block, 0, streams[i]>>>(i, 4);
  }

  ErrorCheck(hipEventRecord(stop, 0)); // 0表示走默认流方式
  ErrorCheck(hipEventElapsedTime(&elapsed_time, start, stop));

  for (int i = 0; i< n_streams; i++) {
    ErrorCheck(cudaStreamDestory(streams[i]));
  }

  free(streams);
  ErrorCheck(cudaEventDestory(start));
  ErrorCheck(cudaEventDestory(stop));
  hipDeviceReset();
  return 0;
}

