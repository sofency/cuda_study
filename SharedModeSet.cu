#include <hip/hip_runtime.h>
#include <stdio.h>
#include "common/common.h"

int main(int argc, char**argv)
{
  int nDeviceNumber = 0;
  // 检查是否支持cuda
  ErrorCheck(hipGetDeviceCount(&nDeviceNumber));
  
  int dev = 0;
  // 设置使用哪个GPU
  ErrorCheck(hipSetDevice(dev));

  hipSharedMemConfig sharedMemConfig;
  // 查询共享模式支持的字节数
  ErrorCheck(hipDeviceGetSharedMemConfig(&sharedMemConfig));
  printf("current shared memory mode: %d\n", sharedMemConfig);

  // 修改模式
  switch (sharedMemConfig) {
    case hipSharedMemBankSizeEightByte: 
      sharedMemConfig = cudaSharedMemBankSizeFouryte;
      ErrorCheck(hipDeviceSetSharedMemConfig(sharedMemConfig));
      break;
    case cudaSharedMemBankSizeFouryte:
      sharedMemConfig = hipSharedMemBankSizeEightByte;
      ErrorCheck(hipDeviceSetSharedMemConfig(sharedMemConfig));
      break;
  }

  printf("current shared memory mode: %d\n", sharedMemConfig);

  // 配置共享内存大小
  hipFuncCache_t cacheConfig;
  // 获取共享内存配置
  ErrorCheck(hipDeviceGetCacheConfig(&cacheConfig));
  printf("default cache config for device: %d\n", cacheConfig);
  
  cacheConfig = hipFuncCachePreferEqual; // L1缓存共享内存使用同样大小
  ErrorCheck(hipDeviceSetCacheConfig(cacheConfig));

  cacheConfig = hipFuncCachePreferShared; //使用比较大的共享内存
  ErrorCheck(hipFuncSetCacheConfig(cacheConfig));

  // 获取共享内存配置
  ErrorCheck(hipDeviceGetCacheConfig(&cacheConfig));
  printf("current cache config for device: %d\n", cacheConfig);



  hipDeviceReset();
  return 0;
}