#include "hip/hip_runtime.h"
#include "common/common.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <omp.h>

/**
 * @brief nvcc -Xcompiler -fopenmp 00-stream_dispatch.cu -o stream_dispatch & ./stream_dispatch 
 * 
 * @param argc 
 * @param argv 
 * @return int 
 */
 
 #define NSTREAM 4
 
 __device__ void kernel_func()
 {
   double sum = 0.0;
   long i = 999999;
   while(i>0) {
     for (long j = 0; j < 999999; j++) {
       sum = sum + tan(0.1) * tan(0.1);
     }
     i = i - 1;
   }
 }
 
 __global__ void kernel(int stream, int id) {
   if (0 == threadIdx.x) {
     printf("kernel %d is executed in stream_%d\n", id, stream);
   }
   kernel_func();
 }
 
 int main(int argc, char**argv) {
   int nDeviceNumber = 0;
   // 检测当前设备与cuda兼容的设备
   ErrorCheck(hipGetDeviceCount(&nDeviceNumber));
   // set up device
   int dev = 0;
   // 设置GPU设备
   ErrorCheck(hipSetDevice(dev));
 
   float elapsed_time;
 
   int n_streams = NSTREAM;
   hipStream_t* streams = (hipStream_t*) malloc(n_streams * sizeof(hipStream_t));
 
   for (int i = 0; i< n_streams; i++) {
     ErrorCheck(hipStreamCreate(streams[i]));
   }
 
   dim3 block(1);
   dim3 grid(1);
 
   hipEvent_t start, stop;
   ErrorCheck(hipEventCreate(&start));
   ErrorCheck(hipEventCreate(&stop));
 
   // record start
   ErrorCheck(hipEventRecord(start, 0)); // 0表示走默认流方式
   
   // 使用openmp多线程分配流
   omp_set_num_threads(NSTREAM);
   #pragma omp parallel{
     // 获取线程id
     int threadId = omp_get_thread_num();
     kernel<<<grid, block, 0, streams[threadId]>>>(threadId, 1);
     kernel<<<grid, block, 0, streams[threadId]>>>(threadId, 2);
     kernel<<<grid, block, 0, streams[threadId]>>>(threadId, 3);
     kernel<<<grid, block, 0, streams[threadId]>>>(threadId, 4);
   }
   
     
   
 
   ErrorCheck(hipEventRecord(stop, 0)); // 0表示走默认流方式
   ErrorCheck(hipEventElapsedTime(&elapsed_time, start, stop));
 
   for (int i = 0; i< n_streams; i++) {
     ErrorCheck(cudaStreamDestory(streams[i]));
   }
 
   free(streams);
   ErrorCheck(cudaEventDestory(start));
   ErrorCheck(cudaEventDestory(stop));
   hipDeviceReset();
   return 0;
 }
 
 