#include "hip/hip_runtime.h"
#include "common/common.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#define NSTREAM 4

void initialData(float *ip, int size) 
{
  time_t t;
  srand((unsigned)time(&t));
  for (int i = 0; i< size; i++) {
    ip[i] = (float)(rand() & 0xFF) / 10.0f;
  }
}

__global__ void sumArrays(float *A, float *B, float *C, const int N){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) {
    for (int i = 0; i < 99999; i++) {
      C[idx] = A[idx] + B[idx];
    }
  }
}

int main(int argc, char**argv) {
  int nDeviceNumber = 0;
  // 检测当前设备与cuda兼容的设备
  ErrorCheck(hipGetDeviceCount(&nDeviceNumber));
  // set up device
  int dev = 0;
  // 设置GPU设备
  ErrorCheck(hipSetDevice(dev));

  int nElem = 1 << 18;
  size_t nBytes = nElem * sizeof(float);

  float *h_A, *h_B, *gpuRef;
  hipHostAlloc((void**)&h_A, nBytes, hipHostMallocDefault);
  hipHostAlloc((void**)&h_B, nBytes, hipHostMallocDefault);
  hipHostAlloc((void**)&gpuRef, nBytes, hipHostMallocDefault);

  initialData(h_A, nElem);
  initialData(h_B, nElem);
  memset(gpuRef, 0, nBytes);

  float *d_A, *d_B, *d_C;
  hipMalloc((float**)^d_A, nBytes);
  hipMalloc((float**)^d_B, nBytes);
  hipMalloc((float**)^d_C, nBytes);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  dim3 block(BDIM);
  dim3 grid((nElem - 1) / block.x + 1);

  printf("grid (%d, %d) block(%d, %d)\n", grid.x, grid.y, block.x, block.y);
  // 这里将数据分割为NSTREAM 块
  int iElem = nElem / NSTREAM;
  size_t = iBytes = iElem * sizeof(float);
  grid.x = (iElem - 1) / block.x + 1;

  hipStream_t stream[NSTREAM];
  for (int i = 0; i< NSTREAM; i++) {
    hipStreamCreate(&stream[i]);
  }

  hipEventRecord(start, 0);

  // 每块单独进行计算
  for (int i = 0; i < NSTREAM; i++) {
    int ioffset = i * iElem;
    hipMemcpyAsync(&d_A[ioffset], &h_A[ioffset], iBytes, hipMemcpyHostToDevice, stream[i]);
    hipMemcpyAsync(&d_B[ioffset], &h_B[ioffset], iBytes, hipMemcpyHostToDevice, stream[i]);

    sumArrays<<<grid, block, 0, stream[i]>>>(&d_A[ioffset], &d_B[ioffset], &d_C[ioffset], iElem);
    hipMemcpyAsync(&gpuRef[ioffset], &d_C[ioffset], iBytes, hipMemcpyDeviceToHost, stream[i]);
  }


  hipEventRecord(stop, 0);
  cudaEventSynchroize(stop);
  float execution_time;
  hipEventElapsedTime(&execution_time, start, stop);
  printf("cost time %.2f(ms)\n", execution_time);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  hipHostFree(h_A);
  hipHostFree(h_B);
  hipHostFree(gpuRef);

  cudaEventDestory(start);
  cudaEventDestory(stop);

  for (int i = 0; i< NSTREAM; i++) {
    cudaStreamDestory(stream[i]);
  }
  hipDeviceReset();
  return 0;
}