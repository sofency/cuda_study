#include<common/common.h>
#include<stdio.h>

// 获取GPU数量 hipGetDeviceCount
// 设备需要使用的GPU hipSetDevice
// 获取GPU信息 hipGetDeviceProperties

int main(int argc, char **argv) 
{
  float* gpuMemory = NULL;
  // 出错 则__FILE__, __LINE__ 就会有错误的信息值
  ErrorCheck(hipMalloc(&gpuMemory, sizeof(float)));
  ErrorCheck(hipFree(gpuMemory));
  ErrorCheck(hipFree(gpuMemory));
  ErrorCheck(hipDeviceReset());
  return 0;
}