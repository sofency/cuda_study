#include "hip/hip_runtime.h"
#include "common/common.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

__global__ void kernel(float *F, double *D) {
  float var1 = *F;
  float var2 = *D;
  printf("device single precision representation is %.2f\n", var1);
  printf("device double precision representation is %.2f\n", var2);
}

int main(int argc, char**argv) {

  float hostF = 0.0;
  double hostD = 0.0;

  if (argc == 2) {
    hostF = (float) atof(argv[1]);
    hostD = (double) atof(aargv[1]);
  } else {
    printf("input a float number\n");
    return -1;
  }

  int nDeviceNumber = 0;
  // 检测当前设备与cuda兼容的设备
  ErrorCheck(hipGetDeviceCount(&nDeviceNumber));
  // set up device
  int dev = 0;
  // 设置GPU设备
  ErrorCheck(hipSetDevice(dev));

  float *deviceF;
  double* deviceD;

  hipMalloc((void**)&deviceF, sizeof(float));
  hipMalloc((void**)&deviceD, sizeof(double));
  hipMemcpy(deviceF, &hostF, sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(deviceD, &hostD, sizeof(double), hipMemcpyHostToDevice);

  printf("host single precision representation is %.2f\n", hostF);
  printf("host double precision representation is %.2f\n", hostD);

  kernel<<<1,1>>>(deviceF, deviceD);
  hipDeviceSynchronize();
  hipFree(deviceF);
  hipFree(deviceD);
  hipDeviceReset();
  return 0;
}